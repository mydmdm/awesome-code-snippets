#include "hip/hip_runtime.h"
#include "Tensors.h"
#include "compute.cuh"
#include "compute.h"
#include "utils_cublas.cuh"

#include "matmul_def.h"

struct TestContext
{
    FILE *_fn;
    PinnedHostAllocator<D> a_host;
    DeviceAllocator<D> a_dev;
    hipblasHandle_t handler;

    Matrix<D, M, K> a;
    Matrix<D, K, N> b;
    Matrix<D, M, N> c0; // ground truth
    // Matrix<D, M, N> c;

    Matrix<D, M, K> d_a;
    Matrix<D, K, N> d_b;
    // Matrix<D, M, N> d_c;

    double flops = 2.0 * M * N * K; // number of MAC (Multiply-Accumulate) operations

    TestContext(const char *fname)
        : a(a_host), b(a_host), c0(a_host),
          d_a(a_dev), d_b(a_dev)
    {
        if (fname)
        {
            _fn = fopen(fname, "a+");
        }
        checkCublasStatus(hipblasCreate(&handler));

        randint<D>(a, 0.0, 100.0);
        randint<D>(b, 0.0, 100.0);
        // set_const<D>(a, 1.0);
        // set_const<D>(b, 1.0);
        copy_memory<D>(&d_a, &a);
        copy_memory<D>(&d_b, &b);

        fprintf(stdout, "generate matrix muliplication test with shape (%lu, %lu, %lu), Gflop = %.2f\n", M, N, K, flops * 1e-9);
        if (flops >= 1e9)
        {
            fprintf(stdout, "too large computation for cpu reference, use cublas for correctness check\n");
            fprintf(stdout, "use a small (M,N,K) for cpu correctness check\n");
            Matrix<D, M, N> d_c(a_dev);
            cublas_matmal<D, M, N, K>(handler, d_a._start, d_b._start, d_c._start);
            // checkCudaStatus(hipDeviceSynchronize());
            copy_memory<D>(&c0, &d_c);
        }
        else
        {
            auto name = "naive_cpu";
            auto now = get_now();
            matmul<D, M, N, K>(a, b, c0);
            auto t = time_difference_ns(now);
            fprintf(stdout, "%s, latency(ns), %e, Gflop/s, %.2f\n", name, (double)t, flops / t);
        }
    }

    ~TestContext()
    {
        checkCublasStatus(hipblasDestroy(handler));
        if (_fn)
        {
            fclose(_fn);
        }
    }

    void verify(const char *name, double t, Matrix<D, M, N> *d_c)
    {
        fprintf(stdout, "name=%s, latency=%.9f, Gflop/s=%.3f\n", t * 1e-9, flops / t);
        if (_fn)
        {
            fprintf(_fn, "name=%s, latency=%.9f, Gflop/s=%.3f\n", t * 1e-9, flops / t);
        }
        Matrix<D, M, N> c(a_host, d_c);
        double max_err;
        auto num_err = count_error(c0, c, &max_err);
        if (num_err)
        {
            fprintf(stdout, "max error is %e\n", max_err);
            fprintf(stdout, "number of error is %lu\n", num_err);
        }
    }

    void test_kernel(const char *name, void (*kernel)(const D *, const D *, D *), dim3 blocks, dim3 threads)
    {
        Matrix<D, M, N> d_c(a_dev);
        auto now = get_now();
        range(i, num_repeat)
        {
            kernel<<<blocks, threads>>>(d_a._start, d_b._start, d_c._start);
        }
        checkCudaStatus(hipDeviceSynchronize());
        auto t = time_difference_ns(now);
        verify(name, t, &d_c);
    }

    void test_cublas()
    {
        Matrix<D, M, N> d_c(a_dev);
        auto now = get_now();
        range(i, num_repeat)
        {
            cublas_matmal<D, M, N, K>(handler, d_a._start, d_b._start, d_c._start);
        }
        checkCudaStatus(hipDeviceSynchronize());
        auto t = time_difference_ns(now);
        verify("cublas", t, &d_c);
    }
};

/* compute C = A*B, A is (M,K), B is (K,N), and C is (M,N)
*/
int main(int argc, char *argv[])
{
    auto prop = print_device_properties();
    fprintf(stdout, "matmul test with shape (M,N,K)=(%d,%d,%d)\n", M, N, K);
    int algo = 0;
    std::string fname = "result.matmul.txt";

    TestContext tc(fname.c_str());

    if (algo == (int)AlgoSel::all || algo == (int)AlgoSel::cublas)
    {
        tc.test_cublas();
    }

    dim3 threads(TILE, TILE);
    dim3 blocks(iceil(N, TILE), iceil(M, TILE));
    fprintf(stdout, "launing kernel blocks=(%u,%u), threads=(%u,%u)\n", blocks.x, blocks.y, threads.x, threads.y);

    if (algo == (int)AlgoSel::all || algo == (int)AlgoSel::naive)
    {
        tc.test_kernel("cu_matmul_naive", cu_matmul_naive<D, M, N, K>, blocks, threads);
    }
    if (algo == (int)AlgoSel::all || algo == (int)AlgoSel::tiled)
    {
        tc.test_kernel("cu_matmul_tiled", cu_matmul_tiled<D, M, N, K, TILE>, blocks, threads);
    }
}