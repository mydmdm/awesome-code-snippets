#include "hip/hip_runtime.h"
#include "Allocator.cuh"
#include "Tensors.h"
#include "compute.h"
#include "utils.h"

#include <stdio.h>

#ifndef N
#define N (1 << 12)
#endif

#ifndef D
#define D float
#endif

// number of threads per block
#ifndef TPB
#define TPB 256
#endif

#ifndef num_repeat
#define num_repeat 100
#endif

/* use template parameter to transfer const parameter and use __restrict__ could help nvcc to optimize
*/
template <typename T, size_t SIZE>
__global__ void cu_vec_add_naive(const T *__restrict__ a, const T *__restrict__ b, T *__restrict__ c)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Boundary check
    if (tid < SIZE)
        c[tid] = a[tid] + b[tid];
}

int main()
{
    auto hoa = PinnedHostAllocator<D>(); // host memory allocator
    auto dva = DeviceAllocator<D>();
    Array<D, N> a(hoa), b(hoa), c0(hoa);
    // randn<D>(a, 0.0, 1.0);
    // randn<D>(b, 0.0, 1.0);
    set_const<D>(a, 1.0);
    set_const<D>(b, 1.0);

    Array<D, N> d_a(dva, &a), d_b(dva, &b), d_c(dva);

    if (1)
    {
        auto now = get_now();
        range(i, num_repeat)
        {
            vector_add<D, N>(a._start, b._start, c0._start);
        }
        auto t = time_difference_ns(now);
        fprintf(stdout, "naive_cpu, %lu\n", t / num_repeat);
        is_const<D>(c0, 2.0);
    }

    if (1)
    {
        auto now = get_now();
        range(i, num_repeat)
        {
            cu_vec_add_naive<D, N><<<iceil(N, TPB), TPB>>>(d_a._start, d_b._start, d_c._start);
        }
        hipDeviceSynchronize();
        auto t = time_difference_ns(now);
        fprintf(stdout, "cu_vec_add_naive, %lu\n", t / num_repeat);
        Array<D, N> c(hoa, &d_c);
        assert_true(memcmp(c0._start, c._start, N * sizeof(D)) == 0, "ComputeWrong");
    }
}
